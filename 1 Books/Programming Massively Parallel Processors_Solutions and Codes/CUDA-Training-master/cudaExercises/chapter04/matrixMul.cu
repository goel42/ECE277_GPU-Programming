/*
 * EXAMPLE OF MATRIX-MATRIX MULTIPLICATION: CHAPTER 4
 */
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>

#define CHECK_ERROR(call) { \
	hipError_t err = call; \
	if (err != hipSuccess) { \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(err); \
	} \
}

__global__
void matrixMulKernel(float *P, float *M, float *N, int Width) {
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    
    // check that only the threads with both Row and Col values are in within range
    if ( Row < Width && Col < Width ) {
        float pValue = 0;
        for(int k = 0; k < Width; k++){
            pValue += M[Row*Width+k] * N[k*Width+Col];
        }
        P[Row*Width+Col] = pValue;
    }
}


void matrixMul(float *h_P, float *h_M, float *h_N, int dim) {
    
    int size = (dim*dim)*sizeof(float);
    float *d_M, *d_N, *d_P;
    
    //1. Allocate global memory on the device for d_Pin and d_Pout
    // With this type of allocation it isn't possible acces using higher-dimensional indexing syntax
    // it need to linearize first.
    CHECK_ERROR(hipMalloc((void**)&d_M, size));
    CHECK_ERROR(hipMalloc((void**)&d_N, size));
    CHECK_ERROR(hipMalloc((void**)&d_P, size));    // assume square matricies
    
    // copy h_Pin to device memory
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    
    //2. Kernel launch code - with 256 threads per block
    dim3 dimGrid(ceil(dim / 16.0),ceil(dim / 16.0),1);
    dim3 dimBlock(16, 16,1);
    matrixMulKernel<<<dimGrid, dimBlock>>>(d_P, d_M, d_N, dim);
    
    //3. copy d_Pout from the device memory
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    
    // Free device vectors
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

int main(int argc, char *argv[]) {
    
    float *h_M, *h_N, *h_P;
    int dim = 1024; // assume square matricies
    
    h_M = (float*)malloc(sizeof(float)*dim*dim);
    h_N = (float*)malloc(sizeof(float)*dim*dim);
    h_P = (float*)malloc(sizeof(float)*dim*dim);
    
    // fill M and N with random float numbers
    srand(time(NULL));
    for (int i = 0; i < dim ; i++) {
        for (int j = 0; j < dim ; j++) {
            h_M[i*dim+j] = ((((float)rand() / (float)(RAND_MAX)) * 10));
            h_N[i*dim+j] = ((((float)rand() / (float)(RAND_MAX)) * 10));
        }
    }
    
    // perform matrix addiction
    matrixMul(h_P, h_M, h_N, dim);
    
    /*********************************************************************************************************
     // verifiy the result
     int valueIsCorrect = 1;
     float mult[dim][dim];
     
     for (int i = 0; i < dim; i++) {
     for (int j = 0; j < dim; j++) {
     mult[i][j] = 0.0;
     }
     }
     
     // Multiplying matrix firstMatrix and secondMatrix and storing in array mult.
     for(int i = 0; i < dim; ++i) {
     for(int j = 0; j < dim; ++j) {
     for(int k = 0; k < dim; ++k) {
     mult[i][j] += h_M[i*dim+k] * h_N[k*dim+j];
     }
     }
     }
     
     for (int i = 0; i < dim && valueIsCorrect; i++) {
     for (int j = 0; j < dim; j++) {
     printf("h_P[%d] != mult[%d][%d] --|-- %f != %f\n", (i*dim+j), i, j, h_P[i*dim+j], mult[i][j]);
     if (h_P[i*dim+j] != mult[i][j]) {
     valueIsCorrect = 0;
     printf("see error above.....\n");
     break;
     }
     }
     }
     ********************************************************************************************************
     * NON HA SENSO VERIFICARE LA CORRETTEZZA DEL RISULTATO SULL'HOST, VEDI 3.2 fino a 6.0 AL SEGUENTE LINK:
     * http://docs.nvidia.com/cuda/floating-point/
     ********************************************************************************************************/
    
    
    // Free host memory
    free(h_M);
    free(h_N);
    free(h_P);
    
    printf("ok multiplication completed with success!\n");
    
    /*
     if (valueIsCorrect) {
     printf("ok multiplication completed with success!\n");
     }
     else printf("somthing was wrong!\n");
     */
    
    return 0;
}
