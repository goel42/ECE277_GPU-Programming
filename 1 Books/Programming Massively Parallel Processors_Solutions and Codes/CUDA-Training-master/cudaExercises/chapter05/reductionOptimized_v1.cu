#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#include <math.h>


#define BLOCK_SIZE 16
hipError_t addWithCuda(int *h_X, int size);


__global__ void partialSumKernel(int *X, int N)
{
	__shared__ int partialSum[2 * BLOCK_SIZE];
	int tx = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tx;
	partialSum[tx] = (i < N) ?  X[i] : 0;
	partialSum[tx + blockDim.x] = 0;

	for (int stride = blockDim.x; stride > 0; stride = stride/2)
	{
		__syncthreads();
		if (tx <= stride) {
			partialSum[tx] += partialSum[tx + stride];
			//printf("tx[%d], bx[%d]: %d + %d\n", tx, blockIdx.x, partialSum[tx], partialSum[tx + stride]);
		}
	}
	if (tx == 0)
		X[blockIdx.x] = partialSum[tx];
}

int main()
{
	int *h_X;
	int size = 32;

	h_X = (int*)malloc(sizeof(int)*size);
	
	// fill the vector with a simple for loop
	for (int i = 0; i < size; i++) {
		h_X[i] = i;
	}

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(h_X, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	
	for (int i = 0; i < ceil(((float)size)/BLOCK_SIZE) ; i++ ) {
		printf("the partial sum result in block %d is: %d\n", i, h_X[i]);
	}
	
	free(h_X);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *h_X, int size)
{
    int *d_X;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers 
    cudaStatus = hipMalloc((void**)&d_X, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

     // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_X, h_X, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    partialSumKernel<<<ceil(((float)size) / BLOCK_SIZE), BLOCK_SIZE>>>(d_X, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(h_X, d_X, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	hipFree(d_X);

Error:
    hipFree(d_X);

    return cudaStatus;
}
