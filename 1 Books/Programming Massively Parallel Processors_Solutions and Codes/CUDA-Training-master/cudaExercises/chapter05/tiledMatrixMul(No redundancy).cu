/*
 * EXAMPLE OF SQUARE MATRIX MULTIPLICATION CHAPTER 4
 */
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>

#define CHECK_ERROR(call) { \
cudaError_t err = call; \
if (err != hipSuccess) { \
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
    exit(err); \
} \
}

#define TILE_WIDTH 16
#define DIM 1024


__global__
void matrixMulKernel(float *P, float *M, float *N) {
    
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH*2];
    
    int tx = threadIdx.x, bx = blockIdx.x;
    int ty = threadIdx.y, by = blockIdx.y;
    
    // identify row and column of the d_P element to work on
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    
    if ( Row < DIM && Col < DIM ) {
        
        float pValue = 0;
        float pValue2 = 0;
        
        // Loop over the d_M and d_N tiles required to compute the d_P element
        for (int ph = 0; ph < DIM/TILE_WIDTH; ph++) {
            
            // Collaborative loading of d_M and d_N tiles n to the shared memory
            Mds[ty][tx] = M[Row * DIM + ph * TILE_WIDTH + tx];
            Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * DIM + Col];
            Nds[ty][tx+TILE_WIDTH] = N[(ph * TILE_WIDTH + ty) * DIM + Col + (DIM/2)];
            
            // printf("ph = %d; block[%d,%d]; thread[%d,%d] --> Nds[0][%d] = %2.2f\n", ph, blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x, tx, Nds[0][tx]);
            __syncthreads();
            
            
            for(int k = 0; k < TILE_WIDTH; k++){
                //printf("ph = %d; block[%d,%d]; thread[%d,%d] --> %2.2f + %2.2f * %2.2f\n", ph, blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x, pValue, Mds[ty][k], Nds[k][tx]);
                pValue  += Mds[ty][k] * Nds[k][tx];
                pValue2 += Mds[ty][k] * Nds[k][tx+TILE_WIDTH];
                
            }
            __syncthreads();
        }
        P[Row*DIM+Col] = pValue;
        P[Row*DIM+Col + (DIM/2)] = pValue2;
    }
}


float matrixMul(float *h_P, float *h_M, float *h_N) {
    
    int size = (DIM*DIM)*sizeof(float); // assume square matricies
    float *d_M, *d_N, *d_P;
    
    //1. Allocate global memory on the device for d_M, d_N and d_P
    // With this type of allocation it isn't possible acces using higher-dimensional indexing syntax
    // it need to linearize first.
    CHECK_ERROR(hipMalloc((void**)&d_M, size));
    CHECK_ERROR(hipMalloc((void**)&d_N, size));
    CHECK_ERROR(hipMalloc((void**)&d_P, size));
    
    // copy h_M and h_N to device memory
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    
    hipEvent_t startTimeCuda, stopTimeCuda;
    hipEventCreate(&startTimeCuda);
    hipEventCreate(&stopTimeCuda);
    
    //2. Kernel launch code - with TILE_WIDTH^2 threads per block
    hipEventRecord(startTimeCuda, 0);
    
    dim3 dimGrid(ceil((DIM/TILE_WIDTH)/2.0), ceil(DIM/TILE_WIDTH), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    matrixMulKernel<<<dimGrid, dimBlock>>>(d_P, d_M, d_N);
    
    hipEventRecord(stopTimeCuda,0);
    
    //3. copy d_P from the device memory
    hipEventSynchronize(stopTimeCuda);
    float msTime;
    hipEventElapsedTime(&msTime, startTimeCuda, stopTimeCuda);
    printf("KernelTime: %f\n", msTime);
 
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    
    // Free device matricies
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
    
    return msTime;
}

void sequentialMM(float* h_M, float* h_N, float* h_C) {
    for (int i = 0; i < DIM; ++i) {
        for (int j = 0; j < DIM; ++j) {
            float sum = 0.0;
            for (int k = 0; k < DIM; ++k)
                sum += h_M[i * DIM + k] * h_N[k * DIM + j];
            h_C[i * DIM + j] = sum;
        }
    }
}

int main(int argc, char *argv[]) {
    
    float *h_M, *h_N, *h_P, *h_C;
    float msTime, msTime_seq;
    hipEvent_t startTimeCuda, stopTimeCuda;
    
    hipEventCreate(&startTimeCuda);
    hipEventCreate(&stopTimeCuda);
    
    h_M = (float*)malloc(sizeof(float)*DIM*DIM);
    h_N = (float*)malloc(sizeof(float)*DIM*DIM);
    h_P = (float*)malloc(sizeof(float)*DIM*DIM);
    h_C = (float*)malloc(sizeof(float)*DIM*DIM);
    
    // fill M and N with float numbers
    srand(time(NULL));
    for (int i = 0; i < DIM ; i++) {
        for (int j = 0; j < DIM ; j++) {
            h_M[i*DIM+j] = ((((float)rand() / (float)(RAND_MAX)) * 10));
            h_N[i*DIM+j] = ((((float)rand() / (float)(RAND_MAX)) * 10));
            h_C[i] = 0.0;
        }
    }
    
    // perform matrix multiplication
    msTime = matrixMul(h_P, h_M, h_N);
    
    // ------- perform matrix multiplication on host ---------
    hipEventRecord(startTimeCuda, 0);
    sequentialMM(h_M, h_N, h_C);
    hipEventRecord(stopTimeCuda,0);
    hipEventSynchronize(stopTimeCuda);
    hipEventElapsedTime(&msTime_seq, startTimeCuda, stopTimeCuda);
    printf("HostTime: %f\n", msTime_seq);
    
    /********************************************************************************************************
     * NON HA SENSO VERIFICARE LA CORRETTEZZA DEL RISULTATO SULL'HOST, VEDI 3.2 fino a 6.0 AL SEGUENTE LINK:
     * http://docs.nvidia.com/cuda/floating-point/
     ********************************************************************************************************
    // verify the result
    for (int i = 0; i < DIM * DIM; ++i) {
        if (h_C[i] != h_P[i]) {
            printf("\x1b[31mError\x1b[0m into result: h_C[%d] = %f != %f = h_P[%d]\n", i, h_C[i], h_P[i], i);
            goto Error;
        }
    }
    */
    
    // Free host memory
    free(h_M);
    free(h_N);
    free(h_P);
    free(h_C);
    
    printf("Ok multiplication completed with \x1b[32msuccess\x1b[0m!\n\n");
    printf("Speedup: %f\n", msTime_seq/msTime);
    
    return 0;
    
Error:
    free(h_M);
    free(h_N);
    free(h_P);
    free(h_C);
    
    return -1;
}
