/*
 * EXAMPLE OF PATTERN CONVOLUTION CHAPTER 7
 * Introducing L2 cache
 */

#include <hip/hip_runtime.h>



#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define CHECK_ERROR(call) { \
cudaError_t err = call; \
if (err != hipSuccess) { \
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
    exit(err); \
    } \
}

#define TILE_SIZE 128
#define MAX_MASK_WIDTH 10
__constant__ float M[MAX_MASK_WIDTH];

// compute vector convolution
// each thread performs one pair-wise convolution
__global__
void convolution_1D_tiled_kernel(float *N, float *P, int Mask_Width, int Width){
    __shared__ float N_ds[TILE_SIZE];
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    N_ds[threadIdx.x] = N[i];
    
    __syncthreads();
    
    int This_tile_start_point = blockIdx.x * blockDim.x;
    int Next_tile_start_point = (blockIdx.x+1) * blockDim.x;
    int N_start_point = i - (Mask_Width/2);
    float Pvalue = 0;
    
    for (int j = 0; j < Mask_Width; j++) {
        int N_index = N_start_point + j;
        // check if we are inner the input array
        if (N_index >= 0 && N_index < Width) {
            // check if we are inner the current block
            if ((N_index >= This_tile_start_point) && N_index < Next_tile_start_point) {
                Pvalue += N_ds[threadIdx.x + j - (Mask_Width/2)] * M[j];
            }
            else{
                Pvalue += N[N_index] * M[j]; // N is hopefully in L2 cache
            }
        }
    }
    P[i] = Pvalue;
}

float convolution_1D_tiled(float *h_N, float *h_M, float *h_P, int Mask_Width, int Width) {
    
    float *d_N, *d_P;
    int sizeWidth = Width*sizeof(float);
    
    hipEvent_t startTimeCuda, stopTimeCuda;
    hipEventCreate(&startTimeCuda);
    hipEventCreate(&stopTimeCuda);
    
    //1. Allocate global memory on the device for N, M and P
    CHECK_ERROR(hipMalloc((void**)&d_N, sizeWidth));
    CHECK_ERROR(hipMalloc((void**)&d_P, sizeWidth));
    
    // copy N to device memory
    hipMemcpy(d_N, h_N, sizeWidth, hipMemcpyHostToDevice);
    
    // Inform CUDA runtime that the data being copied into the constant memory
    // will not be changed during the kernel execution
    hipMemcpyToSymbol(HIP_SYMBOL(M), h_M, Mask_Width*sizeof(float));
    
    //2. Kernel launch code - to have the device to perform the actual convolution
    // ------------------- CUDA COMPUTATION ---------------------------
    hipEventRecord(startTimeCuda, 0);
    
    dim3 dimGrid(ceil((float)Width / (float)TILE_SIZE),1,1);
    dim3 dimBlock(TILE_SIZE,1,1);
    convolution_1D_tiled_kernel<<<dimGrid, dimBlock>>>(d_N, d_P, Mask_Width, Width);
    
    hipEventRecord(stopTimeCuda, 0);
    
    // ---------------------- CUDA ENDING -----------------------------
    hipEventSynchronize(stopTimeCuda);
    float msTime;
    hipEventElapsedTime(&msTime, startTimeCuda, stopTimeCuda);
    printf("KernelTime: %f\n", msTime);
    
    //3. copy C from the device memory
    hipMemcpy(h_P, d_P, sizeWidth, hipMemcpyDeviceToHost);
    
    // Free device vectors
    hipFree(d_N);
    hipFree(d_P);
    
    return msTime;
}

void printArray(float *A, int size){
    for (int i = 0; i < size; i++) {
        printf("%.2f ", A[i]);
    }
    printf("\n");
}

void sequentialConv(float *h_N, float *h_M, float *h_PS, int n, int Mask_Width){
    for (int i = 0, pos; i < n; i++) {
        pos = i - Mask_Width/2;
        for (int j = 0; j < Mask_Width; j++) {
            if (j + pos >= 0 && j + pos < n)
                h_PS[i] += h_N[j + pos] * h_M[j];
        }
    }
}

int main(void) {
    
    // create and host vectors
    float *h_P, *h_N, *h_PS;
    const float val = (float)1/(float)2;
    const int n = 100000;
    const int Mask_Width = 5;
    float h_M[] = {-val, 0, val}; // the mask
    float msTime, msTime_seq;
    hipEvent_t startTimeCuda, stopTimeCuda;
    
    hipEventCreate(&startTimeCuda);
    hipEventCreate(&stopTimeCuda);
    
    // allocate memory for host vectors
    //h_M = (int*)malloc(sizeof(int)*Mask_Width);   // mask array
    h_N = (float*)malloc(sizeof(float)*n);          // input array
    h_P = (float*)malloc(sizeof(float)*n);          // output array
    h_PS = (float*)malloc(sizeof(float)*n);         // output array sequential result
    
    // set initial values for vectors
    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        h_P[i] = 0.0;
        h_PS[i] = 0.0;
        h_N[i] = i + 1;
        //h_N[i] = ((float)rand() / (float)(RAND_MAX)) * 100;
    }
    
    // -------------------------- parrallel convolution -----------------------------------
    msTime = convolution_1D_tiled(h_N, h_M, h_P, Mask_Width, n);
    
    // -------------------------- perform sequential convolution --------------------------
    hipEventRecord(startTimeCuda, 0);
    sequentialConv(h_N, h_M, h_PS, n, Mask_Width);
    hipEventRecord(stopTimeCuda, 0);
    hipEventSynchronize(stopTimeCuda);
    hipEventElapsedTime(&msTime_seq, startTimeCuda, stopTimeCuda);
    printf("HostTime: %f\n", msTime_seq);
    
    /*
     printf("----------------- ARRAY INPUT -----------------\n");
     printArray(h_N, n);
     
     printf("---------- ARRAY RESULT - SEQUENTIAL ----------\n");
     printArray(h_PS, n);
     
     printf("---------- ARRAY RESULT - PARALLEL ------------\n");
     printArray(h_P, n);
     */
    
    
    // verify the result
    for (int i = 0; i < n; i++) {
        if(h_P[i] != h_PS[i]){
            printf("\x1b[31mError\x1b[0m into result: h_P[%d] = %.2f != %.2f = h_PS[%d]\n", i, h_P[i], h_PS[i], i);
            goto Error;
        }
    }
    
    printf("Ok convolution completed with \x1b[32msuccess\x1b[0m!\n\n");
    printf("Speedup: %f\n", msTime_seq/msTime);
    
    // Free host memory
    free(h_N);
    free(h_P);
    free(h_PS);
    
    return 0;
    
Error:
    free(h_N);
    free(h_P);
    free(h_PS);
    return -1;
}
