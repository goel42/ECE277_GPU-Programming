#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
//  EXAMPLE OF INCLUSIVE PREFIX-SCAN CHAPTER 8
//  Brent_Kung_scan
//
////////////////////////////////////////////////////////////////////////////
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//The maximum number of threads is of section elements
#define SECTION_SIZE 2048

hipError_t Brent_Kung_scan(float *X, float *Y, unsigned int size, float *msTime);
void sequential_scan(float *x, float *y, int Max_i);
void print_Array(float *A, int size);
int verify_result(float *Y, float *YS, int size);

////////////////////////////////////////////////////////////////////////////////
//! Simple bad prefix sum 
//! @param X  input data in global memory
//! @param Y  output data in global memory
//! @param InputSize size of input and output data
////////////////////////////////////////////////////////////////////////////////
__global__ void Brent_Kung_scan_kernel(float *X, float *Y, int InputSize)
{
	__shared__ float XY[SECTION_SIZE];
	int i = 2 * blockIdx.x*blockDim.x + threadIdx.x;
	if (i < InputSize) XY[threadIdx.x] = X[i];
	if (i + blockDim.x < InputSize) XY[threadIdx.x + blockDim.x] = X[i + blockDim.x];

	for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
		__syncthreads();
		int index = (threadIdx.x + 1) * 2 * stride - 1;
		if (index < SECTION_SIZE) {
			XY[index] += XY[index - stride];
		}
	}

	for (int stride = SECTION_SIZE / 4; stride > 0; stride /= 2) {
		__syncthreads();
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index + stride < SECTION_SIZE) {
			XY[index + stride] += XY[index];
		}
	}

	__syncthreads();
	if (i < InputSize) Y[i] = XY[threadIdx.x];
	if (i + blockDim.x < InputSize) Y[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
	const int arraySize = 2048;
	float *Y, *YS, *X;
	//float X[arraySize] = { 2,1,3,1,0,4,1,2,0,3,1,2,5,3,1,2 };
	float msTime, msTime_seq;
	hipEvent_t startTimeCuda, stopTimeCuda;
	hipEventCreate(&startTimeCuda);
	hipEventCreate(&stopTimeCuda);

	X = (float*)malloc(arraySize * sizeof(float));
	Y = (float*)malloc(arraySize * sizeof(float));
	YS = (float*)malloc(arraySize * sizeof(float));

	//fill input vector
	for (int i = 0; i < arraySize; i++) {
		X[i] = (float)(i + 1.0);
	}

	//printf("Array input: ");
	//print_Array(X, arraySize);

	// ------------------ Perform sequential scan. -----------------------------
	printf("Sequential scan...\n");
	hipEventRecord(startTimeCuda, 0);
	hipEventSynchronize(startTimeCuda);

	sequential_scan(X, YS, arraySize);

	hipEventRecord(stopTimeCuda, 0);
	hipEventSynchronize(stopTimeCuda);
	hipEventElapsedTime(&msTime_seq, startTimeCuda, stopTimeCuda);
	printf("HostTime: %f\n\n", msTime_seq);
	//printf(" OK!\n");
	//print_Array(YS, arraySize);

	// ------------------ perform parallel scan. -------------------------------
	printf("Parallel scan...\n");
	hipError_t cudaStatus = Brent_Kung_scan(X, Y, arraySize, &msTime);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		goto Error;
	}
	//printf(" OK!\n");
	//print_Array(Y, arraySize);

	// ------------------ verify the result. -----------------------------------
	if (verify_result(Y, YS, arraySize)) {
		goto Error;
	}
	printf("TEST PASSED!\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		goto Error;
	}

	printf("Speedup: %f\n", msTime_seq / msTime);

	free(X);
	free(Y);
	free(YS);
#ifdef WIN32
	system("pause");
#endif // WIN32
	return 0;

Error:
	free(X);
	free(Y);
	free(YS);
#ifdef WIN32
	system("pause");
#endif // WIN32
	return 1;
}

// Helper function for using CUDA to perform scan in parallel.
hipError_t Brent_Kung_scan(float *X, float *Y, unsigned int size, float *msTime)
{
	float *dev_X, *dev_Y;
	hipError_t cudaStatus;
	hipEvent_t startTimeCuda, stopTimeCuda;
	hipEventCreate(&startTimeCuda);
	hipEventCreate(&stopTimeCuda);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for two vectors.
	cudaStatus = hipMalloc((void**)&dev_X, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Y, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vector from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_X, X, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	hipEventRecord(startTimeCuda, 0);
	hipEventSynchronize(startTimeCuda);

	Brent_Kung_scan_kernel << < 1, SECTION_SIZE/2 >> >(dev_X, dev_Y, size);

	hipEventRecord(stopTimeCuda, 0);
	hipEventSynchronize(stopTimeCuda);
	hipEventElapsedTime(msTime, startTimeCuda, stopTimeCuda);
	printf("KernelTime: %f\n\n", *msTime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Y, dev_Y, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_X);
	hipFree(dev_Y);

	return cudaStatus;
}

void sequential_scan(float *x, float *y, int Max_i) {
	float accumulator = x[0];
	y[0] = accumulator;
	for (int i = 1; i < Max_i; i++) {
		accumulator += x[i];
		y[i] = accumulator;
	}
}

void print_Array(float *A, int size) {
	for (int i = 0; i < size; i++) {
		printf("%.2f ", A[i]);
	}
	printf("\n\n");
}

int verify_result(float *Y, float *YS, int size) {
	for (int i = 0; i < size; i++) {
		if (Y[i] != YS[i]) {
			printf("Error Y[%d] = %.2f != %.2f = YS[%d]\n", i, Y[i], YS[i], i);
			return 1;
		}
	}
	return 0;
}
