#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
//  EXAMPLE OF BAD PREFIX-SCAN CHAPTER 8
//  inclusive Kogge_Stone_scan
//
////////////////////////////////////////////////////////////////////////////
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>


//Assumption: the number of threads will be equal to section elements
#define SECTION_SIZE 1024

hipError_t Kogge_Stone_scan(float *X, float *Y, unsigned int size, float *msTime);
void sequential_scan(float *x, float *y, int Max_i);
void print_Array(float *A, int size);
int verify_result(float *Y, float *YS, int size);

////////////////////////////////////////////////////////////////////////////////
//! Simple bad prefix sum 
//! @param X  input data in global memory
//! @param Y  output data in global memory
//! @param InputSize size of input and output data
////////////////////////////////////////////////////////////////////////////////
__global__ void Kogge_Stone_scan_kernel(float *X, float *Y, int InputSize)
{
	__shared__ float XY[SECTION_SIZE];
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < InputSize) {
		XY[threadIdx.x] = X[i];
	}

	// Perform iterative scan on XY
	for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
		float in;
		__syncthreads();
		if (threadIdx.x >= stride){
			in = XY[threadIdx.x - stride];
		}
		__syncthreads();
		if (threadIdx.x >= stride){
			XY[threadIdx.x] += in;
		}
	}

	__syncthreads();
	Y[i] = XY[threadIdx.x];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
	const int arraySize = 1024;
	//float X[arraySize] = { 3, 1, 7, 0, 4, 1, 6, 3 };
	float *Y, *YS, *X;
	float msTime, msTime_seq;
	hipEvent_t startTimeCuda, stopTimeCuda;
	hipEventCreate(&startTimeCuda);
	hipEventCreate(&stopTimeCuda);

	X = (float*)malloc(arraySize * sizeof(float));
	Y = (float*)malloc(arraySize * sizeof(float));
	YS = (float*)malloc(arraySize * sizeof(float));

	//fill input vector
	for (int i = 0; i < arraySize; i++) {
		X[i] = (float)(i + 1.0);
	}

	//printf("Array input: ");
	//print_Array(X, arraySize);

	// ---------------------- PERFORM SEQUENTIAL SCAN ----------------
	printf("Sequential scan...\n");
	hipEventRecord(startTimeCuda, 0);
	hipEventSynchronize(startTimeCuda);

	sequential_scan(X, YS, arraySize);

	hipEventRecord(stopTimeCuda, 0);
	hipEventSynchronize(stopTimeCuda);
	hipEventElapsedTime(&msTime_seq, startTimeCuda, stopTimeCuda);
	printf("HostTime: %f\n\n", msTime_seq);
	//print_Array(YS, arraySize);
	//printf(" OK!\n");

	// ---------------------- PERFORM PARALELL SCAN ------------------
	printf("parallel scan...\n");
	hipError_t cudaStatus = Kogge_Stone_scan(X, Y, arraySize, &msTime);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		goto Error;
	}
	//print_Array(Y, arraySize);
	//printf(" OK!\n");

	// ----------------------- VERIFY THE RESULT ---------------------
	if (verify_result(Y, YS, arraySize)) {
		goto Error;
	}
	printf("TEST PASSED!\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		goto Error;
	}

	printf("Speedup: %f\n", msTime_seq / msTime);

	free(X);
	free(Y);
	free(YS);
#ifdef WIN32
	system("pause");
#endif // WIN32
	return 0;

Error:
	free(X);
	free(Y);
	free(YS);
#ifdef WIN32
	system("pause");
#endif // WIN32
	return 1;
}

// Helper function for using CUDA to perform scan in parallel.
hipError_t Kogge_Stone_scan(float *X, float *Y, unsigned int size, float *msTime)
{
	float *dev_X, *dev_Y;
	hipError_t cudaStatus;
	hipEvent_t startTimeCuda, stopTimeCuda;
	hipEventCreate(&startTimeCuda);
	hipEventCreate(&stopTimeCuda);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for two vectors.
	cudaStatus = hipMalloc((void**)&dev_X, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Y, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vector from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_X, X, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	hipEventRecord(startTimeCuda, 0);
	hipEventSynchronize(startTimeCuda);


	Kogge_Stone_scan_kernel << < 1, SECTION_SIZE >> >(dev_X, dev_Y, size);

	hipEventRecord(stopTimeCuda, 0);
	hipEventSynchronize(stopTimeCuda);
	hipEventElapsedTime(msTime, startTimeCuda, stopTimeCuda);
	printf("KernelTime: %f\n\n", *msTime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Y, dev_Y, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_X);
	hipFree(dev_Y);

	return cudaStatus;
}

void sequential_scan(float *x, float *y, int Max_i) {
	float accumulator = x[0];
	y[0] = accumulator;
	for (int i = 1; i < Max_i; i++) {
		accumulator += x[i];
		y[i] = accumulator;
	}
}

void print_Array(float *A, int size) {
	for (int i = 0; i < size; i++) {
		printf("%.2f ", A[i]);
	}
	printf("\n\n");
}

int verify_result(float *Y, float *YS, int size) {
	for (int i = 0; i < size; i++) {
		if (Y[i] != YS[i]) {
			printf("Error Y[%d] = %.2f != %.2f = YS[%d]\n", i, Y[i], YS[i], i);
			return 1;
		}
	}
	return 0;
}
