#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
//  EXAMPLE OF HIERARCHICAL THREE PHASE PREFIX-SCAN CHAPTER 8
//  Efficient_Kogge_Stone_scan:
//	Using this three-phase approach, we can use a much smaller number of
//	threads then the number of the elements in a section. The maximal size
//	of a section is no longer limited by the number of threads in the block
//	but rather, the size of shared memory; all elements in a section
//	must to fit into the shared memory.
//
////////////////////////////////////////////////////////////////////////////
//	With 8192 elements using float numbers there are approximation problems 
////////////////////////////////////////////////////////////////////////////

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define SECTION_SIZE 4096
#define BLOCK_DIM 1024
#define SUBSECTION_SIZE SECTION_SIZE / BLOCK_DIM

hipError_t efficient_Kogge_Stone_scan(float *X, float *Y, unsigned int size, float *msTime);
void sequential_scan(float *x, float *y, int Max_i);
void print_Array(float *A, int size);
int verify_result(float *Y, float *YS, int size);

__device__
void print_Array_device(float *A, int size) {
	for (int i = 0; i < size; i++) {
		printf("A[%d] = %.2f\n", i, A[i]);
	}
	printf("\n\n");
}

////////////////////////////////////////////////////////////////////////////////
//! Efficient prefix sum 
//! @param X  input data in global memory
//! @param Y  output data in global memory
//! @param InputSize size of input and output data
////////////////////////////////////////////////////////////////////////////////
__global__ void efficient_Kogge_Stone_scan_kernel(float *X, float *Y, int InputSize) {
	__shared__ float XY[SECTION_SIZE];
	__shared__ float AUS[BLOCK_DIM];
	//int i = blockIdx.x * blockDim.x + threadIdx.x;

	// Keep mind: Partition the input into blockDim.x subsections: i.e. for 8 threads --> 8 subsections

	// collaborative load in a coalesced manner
	for (int j = 0; j < SECTION_SIZE; j += blockDim.x) {
		XY[threadIdx.x + j] = X[threadIdx.x + j];
	}
	__syncthreads();


	// PHASE 1: scan inner own subsection
	// At the end of this phase the last element of each subsection contains the sum of all alements in own subsection
	for (int j = 1; j < SUBSECTION_SIZE; j++) {
		XY[threadIdx.x * (SUBSECTION_SIZE)+j] += XY[threadIdx.x * (SUBSECTION_SIZE)+j - 1];
	}
	__syncthreads();


	// PHASE 2: perform iterative kogge_stone_scan of the last elements of each subsections of XY loaded first in AUS
	AUS[threadIdx.x] = XY[threadIdx.x * (SUBSECTION_SIZE)+(SUBSECTION_SIZE)-1];
	float in;
	for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
		__syncthreads();
		if (threadIdx.x >= stride) {
			in = AUS[threadIdx.x - stride];
		}
		__syncthreads();
		if (threadIdx.x >= stride) {
			AUS[threadIdx.x] += in;
		}
	}
	__syncthreads();


	// PHASE 3: each thread adds to its elements the new value of the last element of its predecessor's section
	if (threadIdx.x > 0) {
		for (unsigned int stride = 0; stride < (SUBSECTION_SIZE); stride++) {
			XY[threadIdx.x * (SUBSECTION_SIZE)+stride] += AUS[threadIdx.x - 1];  // <--
		}
	}
	__syncthreads();


	// store the result into output vector
	for (int j = 0; j < SECTION_SIZE; j += blockDim.x) {
		Y[threadIdx.x + j] = XY[threadIdx.x + j];
	}
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
	const int arraySize = 4096;
	float *Y, *YS, *X;
	//float X[arraySize] = { 2,1,3,1,0,4,1,2,0,3,1,2,5,3,1,2 };
	float msTime, msTime_seq;
	hipEvent_t startTimeCuda, stopTimeCuda;
	hipEventCreate(&startTimeCuda);
	hipEventCreate(&stopTimeCuda);

	X = (float*)malloc(arraySize * sizeof(float));
	Y = (float*)malloc(arraySize * sizeof(float));
	YS = (float*)malloc(arraySize * sizeof(float));

	//fill input vector
	for (int i = 0; i < arraySize; i++) {
		X[i] = (float)(i + 1.0);
	}

	//printf("Array input:");
	//print_Array(X, arraySize);

	// ---------------------- PERFORM SEQUENTIAL SCAN ----------------
	printf("Sequential scan...\n");
	hipEventRecord(startTimeCuda, 0);
	hipEventSynchronize(startTimeCuda);

	sequential_scan(X, YS, arraySize);

	hipEventRecord(stopTimeCuda, 0);
	hipEventSynchronize(stopTimeCuda);
	hipEventElapsedTime(&msTime_seq, startTimeCuda, stopTimeCuda);
	printf("HostTime: %f\n\n", msTime_seq);
	//print_Array(YS, arraySize);
	//printf(" OK!\n");

	// ---------------------- PERFORM PARALELL SCAN ------------------
	printf("Parallel scan...\n");
	hipError_t cudaStatus = efficient_Kogge_Stone_scan(X, Y, arraySize, &msTime);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		goto Error;
	}
	//print_Array(Y, arraySize);
	//printf(" OK!\n");

	// ------------------ verify the result. -----------------------------------
	if (verify_result(Y, YS, arraySize)) {
		goto Error;
	}
	printf("TEST PASSED!\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		goto Error;
	}

	printf("Speedup: %f\n", msTime_seq / msTime);

	free(X);
	free(Y);
	free(YS);
#ifdef WIN32
	system("pause");
#endif // WIN32
	return 0;

Error:
	free(X);
	free(Y);
	free(YS);
#ifdef WIN32
	system("pause");
#endif // WIN32
	return 1;
}

// Helper function for using CUDA to perform scan in parallel.
hipError_t efficient_Kogge_Stone_scan(float *X, float *Y, unsigned int size, float *msTime)
{
	float *dev_X, *dev_Y;
	hipError_t cudaStatus;
	hipEvent_t startTimeCuda, stopTimeCuda;
	hipEventCreate(&startTimeCuda);
	hipEventCreate(&stopTimeCuda);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for two vectors.
	cudaStatus = hipMalloc((void**)&dev_X, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_Y, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vector from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_X, X, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with BLOCK_DIM theads per block.
	hipEventRecord(startTimeCuda, 0);
	hipEventSynchronize(startTimeCuda);

	efficient_Kogge_Stone_scan_kernel << <1, BLOCK_DIM >> > (dev_X, dev_Y, size);

	hipEventRecord(stopTimeCuda, 0);
	hipEventSynchronize(stopTimeCuda);
	hipEventElapsedTime(msTime, startTimeCuda, stopTimeCuda);
	printf("KernelTime: %f\n\n", *msTime);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching efficient_Kogge_Stone_scan_kernel Kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Y, dev_Y, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_X);
	hipFree(dev_Y);

	return cudaStatus;
}

void sequential_scan(float *x, float *y, int Max_i) {
	float accumulator = x[0];
	y[0] = accumulator;
	for (int i = 1; i < Max_i; i++) {
		accumulator += x[i];
		y[i] = accumulator;
	}
}

void print_Array(float *A, int size) {
	for (int i = 0; i < size; i++) {
		printf("%.2f ", A[i]);
	}
	printf("\n\n");
}

int verify_result(float *Y, float *YS, int size) {
	for (int i = 0; i < size; i++) {
		if (Y[i] - YS[i] > 1e-5) {
			printf("Error Y[%d] = %.2f != %.2f = YS[%d]\n", i, Y[i], YS[i], i);
			return 1;
		}
	}
	return 0;
}
