#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Example kernels for transposing a rectangular host array using a variety of
 * optimizations, including shared memory, unrolling, and memory padding.
 */

#define BDIMX 16
#define BDIMY 16

#define IPAD 2

void initialData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void printData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%3.0f ", in[i]);
    }

    printf("\n");
    return;
}

void checkResult(float *hostRef, float *gpuRef, const int size)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < size; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                   gpuRef[i]);
            break;
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void transposeHost(float *out, float *in, const int nx, const int ny)
{
    for( int iy = 0; iy < ny; ++iy)
    {
        for( int ix = 0; ix < nx; ++ix)
        {
            out[ix * ny + iy] = in[iy * nx + ix];
        }
    }
}

__global__ void copyGmem(float *out, float *in, const int nx, const int ny)
{
    // matrix coordinate (ix,iy)
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // transpose with boundary test
    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}

__global__ void naiveGmem(float *out, float *in, const int nx, const int ny)
{
    // matrix coordinate (ix,iy)
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // transpose with boundary test
    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}

__global__ void naiveGmemUnroll(float *out, float *in, const int nx,
                                const int ny)
{
    unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int ti = iy * nx + ix;
    unsigned int to = ix * ny + iy;

    if (ix + blockDim.x < nx && iy < ny)
    {
        out[to]                   = in[ti];
        out[to + ny * blockDim.x]   = in[ti + blockDim.x];
    }
}


__global__ void transposeSmemUnroll(float *out, float *in, const int nx,
                                    const int ny)
{
    // static 1D shared memory
    __shared__ float tile[BDIMY * BDIMX * 2];

    // coordinate in original matrix
    unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    unsigned int ix2 = blockIdx.y * blockDim.y + icol;
    unsigned int iy2 = 2 * blockIdx.x * blockDim.x + irow;

    // linear global memory index for transposed matrix
    unsigned int to = iy2 * ny + ix2;

    if (ix + blockDim.x < nx && iy < ny)
    {
        // load two rows from global memory to shared memory
        unsigned int row_idx = 2 * threadIdx.y * blockDim.x + threadIdx.x;
        tile[row_idx]       = in[ti];
        tile[row_idx + BDIMX] = in[ti + BDIMX];

        // thread synchronization
        __syncthreads();

        // store two rows to global memory from two columns of shared memory
        unsigned int col_idx = icol * blockDim.x * 2 + irow;
        out[to] = tile[col_idx];
        out[to + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}

__global__ void transposeSmemUnrollPad(float *out, float *in, const int nx,
                                       const int ny)
{
    // static 1D shared memory with padding
    __shared__ float tile[BDIMY * (BDIMX * 2 + IPAD)];

    // coordinate in original matrix
    unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    unsigned int ix2 = blockIdx.y * blockDim.y + icol;
    unsigned int iy2 = 2 * blockIdx.x * blockDim.x + irow;

    // linear global memory index for transposed matrix
    unsigned int to = iy2 * ny + ix2;

    if (ix + blockDim.x < nx && iy < ny)
    {
        // load two rows from global memory to shared memory
        unsigned int row_idx = threadIdx.y * (blockDim.x * 2 + IPAD) +
            threadIdx.x;
        tile[row_idx]         = in[ti];
        tile[row_idx + BDIMX] = in[ti + BDIMX];

        // thread synchronization
        __syncthreads();

        // store two rows to global memory from two columns of shared memory
        unsigned int col_idx = icol * (blockDim.x * 2 + IPAD) + irow;
        out[to] = tile[col_idx];
        out[to + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}

__global__ void transposeSmemUnrollPadDyn (float *out, float *in, const int nx,
        const int ny)
{
    // dynamic shared memory
    extern __shared__ float tile[];

    unsigned int ix = blockDim.x * blockIdx.x * 2 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix;

    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    unsigned int ix2 = blockDim.y * blockIdx.y + icol;
    unsigned int iy2 = blockDim.x * 2 * blockIdx.x + irow;
    unsigned int to = iy2 * ny + ix2;

    // transpose with boundary test
    if (ix + blockDim.x < nx && iy < ny)
    {
        // load data from global memory to shared memory
        unsigned int row_idx = threadIdx.y * (blockDim.x * 2 + IPAD) +
            threadIdx.x;
        tile[row_idx]       = in[ti];
        tile[row_idx + BDIMX] = in[ti + BDIMX];

        // thread synchronization
        __syncthreads();

        unsigned int col_idx = icol * (blockDim.x * 2 + IPAD) + irow;
        out[to] = tile[col_idx];
        out[to + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}

__global__ void transposeSmem(float *out, float *in, int nx, int ny)
{
    // static shared memory
    __shared__ float tile[BDIMY][BDIMX];

    // coordinate in original matrix
    unsigned int ix, iy, ti, to;
    ix = blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx, irow, icol;
    bidx = threadIdx.y * blockDim.x + threadIdx.x;
    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    ix = blockDim.y * blockIdx.y + icol;
    iy = blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    to = iy * ny + ix;

    // transpose with boundary test
    if (ix < nx && iy < ny)
    {
        // load data from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ti];

        // thread synchronization
        __syncthreads();

        // store data to global memory from shared memory
        out[to] = tile[icol][irow];
    }
}

__global__ void transposeSmemPad(float *out, float *in, int nx, int ny)
{
    // static shared memory with padding
    __shared__ float tile[BDIMY][BDIMX + IPAD];

    // coordinate in original matrix
    unsigned int  ix, iy, ti, to;
    ix = blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx, irow, icol;
    bidx = threadIdx.y * blockDim.x + threadIdx.x;
    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    ix = blockDim.y * blockIdx.y + icol;
    iy = blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    to = iy * ny + ix;

    // transpose with boundary test
    if (ix < nx && iy < ny)
    {
        // load data from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ti];

        // thread synchronization
        __syncthreads();

        // store data to global memory from shared memory
        out[to] = tile[icol][irow];
    }
}

__global__ void transposeSmemDyn(float *out, float *in, int nx, int ny)
{
    // dynamic shared memory
    extern __shared__ float tile[];

    // coordinate in original matrix
    unsigned int  ix, iy, ti, to;
    ix = blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int row_idx, col_idx, irow, icol;
    row_idx = threadIdx.y * blockDim.x + threadIdx.x;
    irow    = row_idx / blockDim.y;
    icol    = row_idx % blockDim.y;
    col_idx = icol * blockDim.x + irow;

    // coordinate in transposed matrix
    ix = blockDim.y * blockIdx.y + icol;
    iy = blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    to = iy * ny + ix;

    // transpose with boundary test
    if (ix < nx && iy < ny)
    {
        // load data from global memory to shared memory
        tile[row_idx] = in[ti];

        // thread synchronization
        __syncthreads();

        // store data to global memory from shared memory
        out[to] = tile[col_idx];
    }
}

__global__ void transposeSmemPadDyn(float *out, float *in, int nx, int ny)
{
    // static shared memory with padding
    extern __shared__ float tile[];

    // coordinate in original matrix
    unsigned int  ix, iy, ti, to;
    ix = blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;
    unsigned int irow    = idx / blockDim.y;
    unsigned int icol    = idx % blockDim.y;
    unsigned int col_idx = icol * (blockDim.x + IPAD) + irow;

    // coordinate in transposed matrix
    ix = blockDim.y * blockIdx.y + icol;
    iy = blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    to = iy * ny + ix;

    // transpose with boundary test
    if (ix < nx && iy < ny)
    {
        // load data from global memory to shared memory
        tile[row_idx] = in[ti];

        // thread synchronization
        __syncthreads();

        // store data to global memory from shared memory
        out[to] = tile[col_idx];
    }
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting transpose at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool iprint = 0;

    // set up array size 2048
    int nx = 1 << 12;
    int ny = 1 << 12;

    if (argc > 1) iprint = atoi(argv[1]);

    if (argc > 2) nx  = atoi(argv[2]);

    if (argc > 3) ny  = atoi(argv[3]);

    printf(" with matrix nx %d ny %d\n", nx, ny);
    size_t nBytes = nx * ny * sizeof(float);

    // execution configuration
    dim3 block (BDIMX, BDIMY);
    dim3 grid  ((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    dim3 grid2  ((nx + block.x * 2 - 1) / (block.x * 2),
                 (ny + block.y - 1) / block.y);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    //  initialize host array
    initialData(h_A, nx * ny);

    //  transpose at host side
    transposeHost(hostRef, h_A, nx, ny);

    // allocate device memory
    float *d_A, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

    // tranpose gmem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    double iStart = seconds();
    copyGmem<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    float ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) /
        iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("copyGmem elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", iElaps, grid.x, grid.y, block.x,
           block.y, ibnd);

    // tranpose gmem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    naiveGmem<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("naiveGmem elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", iElaps, grid.x, grid.y, block.x,
           block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    naiveGmemUnroll<<<grid2, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    checkResult(hostRef, gpuRef, nx * ny);
    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("naiveGmemUnroll elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", iElaps, grid2.x, grid2.y, block.x,
           block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    transposeSmem<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    checkResult(hostRef, gpuRef, nx * ny);
    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("transposeSmem elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", iElaps, grid.x, grid.y, block.x,
           block.y, ibnd);

    // tranpose smem pad
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    transposeSmemPad<<<grid, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    checkResult(hostRef, gpuRef, nx * ny );
    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("transposeSmemPad elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", iElaps, grid.x, grid.y, block.x,
           block.y, ibnd);

    // tranpose smem pad
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    transposeSmemDyn<<<grid, block, BDIMX*BDIMY*sizeof(float)>>>(d_C, d_A, nx,
            ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    checkResult(hostRef, gpuRef, nx * ny);
    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("transposeSmemDyn elapsed %f sec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", iElaps, grid.x, grid.y, block.x,
           block.y, ibnd);

    // tranpose smem pad
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
  transposeSmemPadDyn<<<grid, block, (BDIMX + IPAD) * BDIMY * sizeof(float)>>>(
          d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    checkResult(hostRef, gpuRef, nx * ny );
    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("transposeSmemPadDyn elapsed %f sec <<< grid (%d,%d) block "
           "(%d,%d)>>> effective bandwidth %f GB\n", iElaps, grid.x, grid.y,
           block.x, block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    transposeSmemUnroll<<<grid2, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    checkResult(hostRef, gpuRef, nx * ny );
    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("transposeSmemUnroll elapsed %f sec <<< grid (%d,%d) block "
           "(%d,%d)>>> effective bandwidth %f GB\n", iElaps, grid2.x, grid2.y,
           block.x, block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    transposeSmemUnrollPad<<<grid2, block>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    checkResult(hostRef, gpuRef, nx * ny );
    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("transposeSmemUnrollPad elapsed %f sec <<< grid (%d,%d) block "
           "(%d,%d)>>> effective bandwidth %f GB\n", iElaps, grid2.x, grid2.y,
           block.x, block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

    iStart = seconds();
    transposeSmemUnrollPadDyn<<<grid2, block, (BDIMX * 2 + IPAD) * BDIMY *
        sizeof(float)>>>(d_C, d_A, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nx * ny);

    checkResult(hostRef, gpuRef, nx * ny );
    ibnd = 2 * nx * ny * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / iElaps;
    ibnd = 2 * nx * ny * sizeof(float) / 1e9 / iElaps;
    printf("transposeSmemUnrollPadDyn elapsed %f sec <<< grid (%d,%d) block "
           "(%d,%d)>>> effective bandwidth %f GB\n", iElaps, grid2.x, grid2.y,
           block.x, block.y, ibnd);

    // free host and device memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_C));
    free(h_A);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
