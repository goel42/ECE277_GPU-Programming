#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int device_C;

__global__ void addkernel(int A, int B)
{
	device_C = A + B;
}

int main(int argc, char **argv)
{
	int host_C;

	addkernel << <1, 1 >> > (2, 3);

	hipMemcpyFromSymbol(&host_C, HIP_SYMBOL(device_C), sizeof(int), 0, hipMemcpyDeviceToHost);
	//hipMemcpy(host_C, C, sizeof(int), hipMemcpyDeviceToHost);

	printf("C=%d\n", host_C);
}

