#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/*
 * This example demonstrates two techniques for using the cuRAND host and device
 * API to generate random numbers for CUDA kernels to consume.
 */

int threads_per_block = 256;
int blocks_per_grid = 30;


/*
 * device_api_kernel uses the cuRAND device API to generate random numbers
 * on-the-fly on the GPU, and then performs some dummy computation using them.
 */
__global__ void device_api_kernel(hiprandState *states, float *out, int N)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    hiprandState *state = states + tid;

    hiprand_init(9384, tid, 0, state);
	//hiprand_init(clock() + tid, tid, 0, state);

    for (i = tid; i < N; i += nthreads)
    {
        float rand = hiprand_uniform(state);
        rand = rand * 2;
        out[i] = rand;
    }
}

/*
 * use_device_api is an examples usage of the cuRAND device API to use the GPU
 * to generate random values on the fly from inside a CUDA kernel.
 */
void use_device_api(int N)
{
    int i;
    static hiprandState *states = NULL;
    float *dOut, *hOut;

    /*
     * Allocate device memory to store the output and cuRAND device state
     * objects (which are analogous to handles, but on the GPU).
     */
    CHECK(hipMalloc((void **)&dOut, sizeof(float) * N));
    CHECK(hipMalloc((void **)&states, sizeof(hiprandState) *
                threads_per_block * blocks_per_grid));
    hOut = (float *)malloc(sizeof(float) * N);

    // Execute a kernel that generates and consumes its own random numbers
    device_api_kernel<<<blocks_per_grid, threads_per_block>>>(states, dOut, N);

    // Retrieve the results
    CHECK(hipMemcpy(hOut, dOut, sizeof(float) * N, hipMemcpyDeviceToHost));

    printf("Sampling of output from device API:\n");

    for (i = 0; i < 10; i++)
    {
        printf("%2.4f\n", hOut[i]);
    }

    printf("...\n");

    free(hOut);
    CHECK(hipFree(dOut));
    CHECK(hipFree(states));
}

int main(int argc, char **argv)
{
    int N = 8388608;

    use_device_api(N);

    return 0;
}
