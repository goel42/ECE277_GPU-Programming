#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
/*
 * Example kernels for transposing a rectangular host array using a variety of
 * optimizations, including shared memory, unrolling, and memory padding.
 */

// Some kernels assume square blocks
#define BDIMX 16
#define BDIMY BDIMX

#define INDEX(ROW, COL, INNER) ((ROW) * (INNER) + (COL))

#define IPAD 1

void initialData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void printData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%3.0f ", in[i]);
    }

    printf("\n");
    return;
}

void checkResult(float *hostRef, float *gpuRef, int rows, int cols)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            int index = INDEX(i, j, cols);
            if (abs(hostRef[index] - gpuRef[index]) > epsilon) {
                match = 0;
                printf("different on (%d, %d) (offset=%d) element in "
                        "transposed matrix: host %f gpu %f\n", i, j, index,
                        hostRef[index], gpuRef[index]);
                break;
            }
        }
        if (!match) break;
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void transposeHost(float *out, float *in, const int nrows, const int ncols)
{
    for (int iy = 0; iy < nrows; ++iy)
    {
        for (int ix = 0; ix < ncols; ++ix)
        {
            out[INDEX(ix, iy, nrows)] = in[INDEX(iy, ix, ncols)];
        }
    }
}

__global__ void copyGmem(float *out, float *in, const int nrows, const int ncols)
{
    // matrix coordinate (ix,iy)
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    // transpose with boundary test
    if (row < nrows && col < ncols)
    {
		    // NOTE this is a transpose, not a copy
        out[INDEX(row, col, nrows)] = in[INDEX(row, col, ncols)];
    }
}

__global__ void naiveGmem(float *out, float *in, const int nrows, const int ncols)
{
    // matrix coordinate (ix,iy)
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    // transpose with boundary test
    if (row < nrows && col < ncols)
    {
        out[INDEX(col, row, nrows)] = in[INDEX(row, col, ncols)];
    }
}

__global__ void naiveGmemUnroll(float *out, float *in, const int nrows,
                                const int ncols)
{
    // Pretend there are twice as many blocks in the x direction
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = (2 * blockIdx.x * blockDim.x) + threadIdx.x;

    if (row < nrows)
    {
        if (col < ncols)
        {
            out[INDEX(col, row, nrows)] = in[INDEX(row, col, ncols)];
        }

        col += blockDim.x;

        if (col < ncols)
        {
            out[INDEX(col, row, nrows)] = in[INDEX(row, col, ncols)];
        }
    }
}

__global__ void transposeSmem(float *out, float *in, int nrows, int ncols)
{
    // static shared memory
    __shared__ float tile[BDIMY][BDIMX];

    // coordinate in original matrix
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

    // linear global memory index for original matrix
    unsigned int offset = INDEX(row, col, ncols);

    if (row < nrows && col < ncols)
    {
      // load data from global memory to shared memory
      tile[threadIdx.y][threadIdx.x] = in[offset];
    }

    // thread index in transposed block
    unsigned int bidx, irow, icol;
    bidx = threadIdx.y * blockDim.x + threadIdx.x;
    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

	  // NOTE - need to transpose row and col on block and thread-block level:
	  // 1. swap blocks x-y
	  // 2. swap thread x-y assignment (irow and icol calculations above)
	  // note col still has continuous threadIdx.x -> coalesced gst
	  col = blockIdx.y * blockDim.y + icol;
	  row = blockIdx.x * blockDim.x + irow;

    // linear global memory index for transposed matrix
	  // NOTE nrows is stride of result, row and col are transposed
    unsigned int transposed_offset = INDEX(row, col, nrows);
    // thread synchronization
    __syncthreads();

	  // NOTE invert sizes for write check
    if (row < ncols && col < nrows)
    {
        // store data to global memory from shared memory
        out[transposed_offset] = tile[icol][irow]; // NOTE icol,irow not irow,icol
    }
}

__global__ void transposeSmemUnroll(float *out, float *in, const int nrows, 
                                            const int ncols) 
{
    // static 1D shared memory
    __shared__ float tile[BDIMY][BDIMX * 2];

    // coordinate in original matrix
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = (2 * blockIdx.x * blockDim.x) + threadIdx.x;

    unsigned int row2 = row;
    unsigned int col2 = col + blockDim.x;

    // linear global memory index for original matrix
    unsigned int offset = INDEX(row, col, ncols);
    unsigned int offset2 = INDEX(row2, col2, ncols);

    // thread index in transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // linear global memory index for transposed matrix
    unsigned int transposed_offset = INDEX(col, row, nrows);
    unsigned int transposed_offset2 = INDEX(col2, row2, nrows);

    if (row < nrows && col < ncols)
    {
        tile[threadIdx.y][threadIdx.x] = in[offset];
    }
    if (row2 < nrows && col2 < ncols)
    {
        tile[threadIdx.y][blockDim.x + threadIdx.x] = in[offset2];
    }

    __syncthreads();

    if (row < nrows && col < ncols)
    {
        out[transposed_offset] = tile[irow][icol];
    }
    if (row2 < nrows && col2 < ncols)
    {
        out[transposed_offset2] = tile[irow][blockDim.x + icol];
    }
}

__global__ void transposeSmemUnrollPad(float *out, float *in, const int nrows,
                                       const int ncols)
{
    // static 1D shared memory with padding
    __shared__ float tile[BDIMY][BDIMX * 2 + IPAD];

    // coordinate in original matrix
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = (2 * blockIdx.x * blockDim.x) + threadIdx.x;

    unsigned int row2 = row;
    unsigned int col2 = col + blockDim.x;

    // linear global memory index for original matrix
    unsigned int offset = INDEX(row, col, ncols);
    unsigned int offset2 = INDEX(row2, col2, ncols);

    // thread index in transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // linear global memory index for transposed matrix
    unsigned int transposed_offset = INDEX(col, row, nrows);
    unsigned int transposed_offset2 = INDEX(col2, row2, nrows);

    if (row < nrows && col < ncols)
    {
        tile[threadIdx.y][threadIdx.x] = in[offset];
    }
    if (row2 < nrows && col2 < ncols)
    {
        tile[threadIdx.y][blockDim.x + threadIdx.x] = in[offset2];
    }

    __syncthreads();

    if (row < nrows && col < ncols)
    {
        out[transposed_offset] = tile[irow][icol];
    }
    if (row2 < nrows && col2 < ncols)
    {
        out[transposed_offset2] = tile[irow][blockDim.x + icol];
    }
}

__global__ void transposeSmemUnrollPadDyn (float *out, float *in, const int nrows,
        const int ncols)
{
    // dynamic shared memory
    extern __shared__ float tile[];

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = (2 * blockIdx.x * blockDim.x) + threadIdx.x;

    unsigned int row2 = row;
    unsigned int col2 = col + blockDim.x;

    // linear global memory index for original matrix
    unsigned int offset = INDEX(row, col, ncols);
    unsigned int offset2 = INDEX(row2, col2, ncols);

    // thread index in transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    unsigned int transposed_offset = INDEX(col, row, nrows);
    unsigned int transposed_offset2 = INDEX(col2, row2, nrows);

    if (row < nrows && col < ncols)
    {
        tile[INDEX(threadIdx.y, threadIdx.x, BDIMX * 2 + IPAD)] = in[offset];
    }
    if (row2 < nrows && col2 < ncols)
    {
        tile[INDEX(threadIdx.y, blockDim.x + threadIdx.x, BDIMX * 2 + IPAD)] =
            in[offset2];
    }

    __syncthreads();

    if (row < nrows && col < ncols)
    {
        out[transposed_offset] = tile[INDEX(irow, icol, BDIMX * 2 + IPAD)];
    }
    if (row2 < nrows && col2 < ncols)
    {
        out[transposed_offset2] = tile[INDEX(irow, blockDim.x + icol, BDIMX * 2 + IPAD)];
    }
}

__global__ void transposeSmemPad(float *out, float *in, int nrows, int ncols)
{
	// static shared memory
	__shared__ float tile[BDIMY][BDIMX+1];

	// coordinate in original matrix
	unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

	// linear global memory index for original matrix
	unsigned int offset = INDEX(row, col, ncols);

	if (row < nrows && col < ncols)
	{
		// load data from global memory to shared memory
		tile[threadIdx.y][threadIdx.x] = in[offset];
	}

	// thread index in transposed block
	unsigned int bidx, irow, icol;
	bidx = threadIdx.y * blockDim.x + threadIdx.x;
	irow = bidx / blockDim.y;
	icol = bidx % blockDim.y;

	// NOTE - need to transpose row and col on block and thread-block level:
	// 1. swap blocks x-y
	// 2. swap thread x-y assignment (irow and icol calculations above)
	// note col still has continuous threadIdx.x -> coalesced gst
	col = blockIdx.y * blockDim.y + icol;
	row = blockIdx.x * blockDim.x + irow;

	// linear global memory index for transposed matrix
	// NOTE nrows is stride of result, row and col are transposed
	unsigned int transposed_offset = INDEX(row, col, nrows);
	// thread synchronization
	__syncthreads();

	// NOTE invert sizes for write check
	if (row < ncols && col < nrows)
	{
		// store data to global memory from shared memory
		out[transposed_offset] = tile[icol][irow]; // NOTE icol,irow not irow,icol
	}
}

__global__ void transposeSmemPadnc(float *out, float *in, int nrows, int ncols)
{
	// static shared memory
	int stride = 33;
	__shared__ float tile[8][32+2];

	// coordinate in original matrix
	unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

	// linear global memory index for original matrix
	unsigned int offset = INDEX(row, col, ncols);

	unsigned int shx = (threadIdx.y & 0x1)*blockDim.x + threadIdx.x;
	unsigned int shy = threadIdx.y >> 1;

	if (row < nrows && col < ncols)
	{
		// load data from global memory to shared memory
		tile[shy][shx] = in[offset];
	}

	// thread index in transposed block
	unsigned int bidx, irow, icol;
	bidx = threadIdx.y * blockDim.x + threadIdx.x;
	irow = bidx / blockDim.y;
	icol = bidx % blockDim.y;

	// NOTE - need to transpose row and col on block and thread-block level:
	// 1. swap blocks x-y
	// 2. swap thread x-y assignment (irow and icol calculations above)
	// note col still has continuous threadIdx.x -> coalesced gst
	col = blockIdx.y * blockDim.y + icol;
	row = blockIdx.x * blockDim.x + irow;

	// linear global memory index for transposed matrix
	// NOTE nrows is stride of result, row and col are transposed
	unsigned int transposed_offset = INDEX(row, col, nrows);
	// thread synchronization
	__syncthreads();

	// NOTE invert sizes for write check
	if (row < ncols && col < nrows)
	{
		// store data to global memory from shared memory
		out[transposed_offset] = tile[icol>>1][(icol& 0x1)*blockDim.y + irow]; // NOTE icol,irow not irow,icol
	}
}

//__global__ void transposeSmemPad(float *out, float *in, int nrows, int ncols)
//{
//    // static shared memory with padding
//    __shared__ float tile[BDIMY][BDIMX + IPAD];
//
//    // coordinate in original matrix
//    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
//    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
//
//    // linear global memory index for original matrix
//    unsigned int offset = INDEX(row, col, ncols);
//
//    // thread index in transposed block
//    unsigned int bidx, irow, icol;
//    bidx = threadIdx.y * blockDim.x + threadIdx.x;
//    irow = bidx / blockDim.y;
//    icol = bidx % blockDim.y;
//
//    // linear global memory index for transposed matrix
//    unsigned int transposed_offset = INDEX(col, row, nrows);
//
//    // transpose with boundary test
//    if (row < nrows && col < ncols)
//    {
//        // load data from global memory to shared memory
//        tile[threadIdx.y][threadIdx.x] = in[offset];
//
//        // thread synchronization
//        __syncthreads();
//
//        // store data to global memory from shared memory
//        out[transposed_offset] = tile[irow][icol];
//    }
//}

__global__ void transposeSmemDyn(float *out, float *in, int nrows, int ncols)
{
    // dynamic shared memory
    extern __shared__ float tile[];

    // coordinate in original matrix
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

    // linear global memory index for original matrix
    unsigned int offset = INDEX(row, col, ncols);

    // thread index in transposed block
    unsigned int row_idx, col_idx, irow, icol;
    row_idx = threadIdx.y * blockDim.x + threadIdx.x;
    irow    = row_idx / blockDim.y;
    icol    = row_idx % blockDim.y;
    col_idx = irow * blockDim.x + icol;

    // linear global memory index for transposed matrix
    unsigned int transposed_offset = INDEX(col, row, nrows);

    // transpose with boundary test
    if (row < nrows && col < ncols)
    {
        // load data from global memory to shared memory
        tile[row_idx] = in[offset];

        // thread synchronization
        __syncthreads();

        // store data to global memory from shared memory
        out[transposed_offset] = tile[col_idx];
    }
}

__global__ void transposeSmemPadDyn(float *out, float *in, int nrows, int ncols)
{
    // static shared memory with padding
    extern __shared__ float tile[];

    // coordinate in original matrix
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

    // linear global memory index for original matrix
    unsigned int offset = INDEX(row, col, ncols);

    // thread index in transposed block
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;
    unsigned int irow    = idx / blockDim.y;
    unsigned int icol    = idx % blockDim.y;
    unsigned int col_idx = irow * (blockDim.x + IPAD) + icol;

    // linear global memory index for transposed matrix
    unsigned int transposed_offset = INDEX(col, row, nrows);

    // transpose with boundary test
    if (row < nrows && col < ncols)
    {
        // load data from global memory to shared memory
        tile[row_idx] = in[offset];

        // thread synchronization
        __syncthreads();

        // store data to global memory from shared memory
        out[transposed_offset] = tile[col_idx];
    }
}

#define NUM_REPS  100

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
	//StopWatchInterface *timer = NULL;
	hipEvent_t start, stop;
	float elapsedTimeInMs = 0.0f;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting transpose at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
	//sdkCreateTimer(&timer);
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

    bool iprint = 0;

    // set up array size 2048
    int nrows = 1 << 12;
    int ncols = 1 << 12;

    if (argc > 1) iprint = atoi(argv[1]);

    if (argc > 2) nrows = atoi(argv[2]);

    if (argc > 3) ncols = atoi(argv[3]);

    printf(" with matrix nrows %d ncols %d\n", nrows, ncols);
    size_t ncells = nrows * ncols;
    size_t nBytes = ncells * sizeof(float);

    // execution configuration
    dim3 block (BDIMX, BDIMY);
    /*
     * Map CUDA blocks/threads to output space. Map rows in output to same
     * x-value in CUDA, columns to same y-value.
     */
    dim3 grid ((ncols + block.x - 1) / block.x, (nrows + block.y - 1) / block.y);
    dim3 grid2 ((grid.x + 2 - 1) / 2, grid.y);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    //  initialize host array
    initialData(h_A, nrows * ncols);

    //  transpose at host side
    transposeHost(hostRef, h_A, nrows, ncols);

    // allocate device memory
    float *d_A, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

    // tranpose gmem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	//sdkStartTimer(&timer);
	checkCudaErrors(hipEventRecord(start, 0));
	for (int i = 0; i < NUM_REPS; i++)
		copyGmem<<<grid, block >>>(d_C, d_A, nrows, ncols);

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipDeviceSynchronize());
	//get the total elapsed time in ms
	//sdkStopTimer(&timer);
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, nrows * ncols);

    float ibnd = 1000*2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) /
		elapsedTimeInMs;
    ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / (elapsedTimeInMs/ NUM_REPS);
    printf("copyGmem elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", elapsedTimeInMs, grid.x, grid.y, block.x,
           block.y, ibnd);

    // tranpose gmem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
	for (int i = 0; i < NUM_REPS; i++)
		naiveGmem<<<grid, block>>>(d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
    CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
    ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
    ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / (elapsedTimeInMs / NUM_REPS);
    printf("naiveGmem elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", elapsedTimeInMs, grid.x, grid.y, block.x,
           block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
    naiveGmemUnroll<<<grid2, block>>>(d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
    CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
    ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
    ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / elapsedTimeInMs;
    printf("naiveGmemUnroll elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
           "effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
           block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
	
	for (int i = 0; i < NUM_REPS; i++)
    transposeSmem<<<grid, block>>>(d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
    CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
	ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
	ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / (elapsedTimeInMs / NUM_REPS);
	printf("smem elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
		"effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
		block.y, ibnd);

    // tranpose smem pad
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
	for (int i = 0; i < NUM_REPS; i++)
		transposeSmemPad<<<grid, block>>>(d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
    CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
	ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
	ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / (elapsedTimeInMs / NUM_REPS);
	printf("smem pad elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
		"effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
		block.y, ibnd);

	// tranpose smem pad (nc)
	CHECK(hipMemset(d_C, 0, nBytes));
	memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
	for (int i = 0; i < NUM_REPS; i++)
		transposeSmemPadnc << <grid, block >> >(d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
	CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

	if (iprint) printData(gpuRef, ncells);

	checkResult(hostRef, gpuRef, ncols, nrows);
	ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
	ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / (elapsedTimeInMs / NUM_REPS);
	printf("smem pad (nc) elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
		"effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
		block.y, ibnd);

    // tranpose smem pad
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
    transposeSmemDyn<<<grid, block, BDIMX*BDIMY*sizeof(float)>>>(d_C, d_A, nrows,
            ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
    CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
	ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
	ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / elapsedTimeInMs;
	printf("transposeSmemDyn elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
		"effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
		block.y, ibnd);

    // tranpose smem pad
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
    transposeSmemPadDyn<<<grid, block, (BDIMX + IPAD) * BDIMY * sizeof(float)>>>(
          d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
	CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
	ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
	ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / elapsedTimeInMs;
	printf("transposeSmemPadDyn elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
		"effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
		block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
    transposeSmemUnroll<<<grid2, block>>>(d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
	CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
	ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
	ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / elapsedTimeInMs;
	printf("transposeSmemUnroll elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
		"effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
		block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
    transposeSmemUnrollPad<<<grid2, block>>>(d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
	CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
	ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
	ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / elapsedTimeInMs;
	printf("transposeSmemUnrollPad elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
		"effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
		block.y, ibnd);

    // tranpose smem
    CHECK(hipMemset(d_C, 0, nBytes));
    memset(gpuRef, 0, nBytes);

	checkCudaErrors(hipEventRecord(start, 0));
    transposeSmemUnrollPadDyn<<<grid2, block, (BDIMX * 2 + IPAD) * BDIMY *
        sizeof(float)>>>(d_C, d_A, nrows, ncols);
	checkCudaErrors(hipEventRecord(stop, 0));
	CHECK(hipDeviceSynchronize());
	checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    if(iprint) printData(gpuRef, ncells);

    checkResult(hostRef, gpuRef, ncols, nrows);
	ibnd = 1000 * 2 * ncells * sizeof(float) / (1024.0 * 1024.0 * 1024.0) / elapsedTimeInMs;
	ibnd = 1000 * 2 * ncells * sizeof(float) / 1e9 / elapsedTimeInMs;
	printf("transposeSmemUnrollPadDyn elapsed %f msec <<< grid (%d,%d) block (%d,%d)>>> "
		"effective bandwidth %f GB\n", elapsedTimeInMs, grid2.x, grid2.y, block.x,
		block.y, ibnd);

    // free host and device memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_C));
    free(h_A);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
