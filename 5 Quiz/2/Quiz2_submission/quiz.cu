#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <stdio.h>


void initialData(float *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF ) / 10.0f;
    }

    return;
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx,
                     const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];

        }

        ia += nx;
        ib += nx;
        ic += nx;
    }

    return;
}


void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("%d: host %f gpu %f\n", i, hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("PASS\n\n");
    else
        printf("FAIL\n\n");
}


__global__ void sumMatrixOnGPU(float *MatA, float *MatB, float *MatC, int nx,
                                 int ny)
{
  // Program kernel codes properly, otherwise your system could crash /* FIXME */
	// int ix, iy, idx; 
	unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idx = iy * nx + ix;
	
	if ((ix < nx) && (iy < ny)) {
		MatC[idx] = MatA[idx] + MatB[idx];
	}

}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    checkCudaErrors(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    initialData(h_A, nxy);
    initialData(h_B, nxy);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    checkCudaErrors(hipMalloc((void **)&d_MatA, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_MatB, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_MatC, nBytes));

    // transfer data from host to device
    checkCudaErrors(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int dimx = 16;
	int dimy = 16;

	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    sumMatrixOnGPU<<<grid, block >>>(d_MatA, d_MatB, d_MatC, nx, ny);

    // checkCudaErrors kernel error
    checkCudaErrors(hipGetLastError());

    // copy kernel result back to host side
    checkCudaErrors(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    // checkCudaErrors device results
    checkResult(hostRef, gpuRef, nxy);

	printf("Haenara Shin, A53233226, #29\n");

    // free device global memory
    checkCudaErrors(hipFree(d_MatA));
    checkCudaErrors(hipFree(d_MatB));
    checkCudaErrors(hipFree(d_MatC));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    checkCudaErrors(hipDeviceReset());

    return (0);
}
